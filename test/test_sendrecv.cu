#include "hip/hip_runtime.h"
#include "../src/sendrecv.cu"
#include "../src/channel.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cuda/atomic>
#include <chrono>
#include <thread>
#include <hip/hip_runtime.h>
#include <atomic>

// Helper function to check CUDA errors
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)

// Test data size
const size_t TEST_DATA_SIZE = 1024 * 1024;  // 1MB test data

// Initialize test data on GPU
__global__ void initTestData(uint64_t* data, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size / sizeof(uint64_t)) {
        data[idx] = idx;  // Fill with index values
    }
}

// Verify data on CPU
bool verifyData(uint64_t* data, size_t size) {
    for (size_t i = 0; i < size / sizeof(uint64_t); i++) {
        if (data[i] != i) {
            std::cerr << "Data mismatch at index " << i 
                      << ": expected " << i << ", got " << data[i] << std::endl;
            return false;
        }
    }
    return true;
}

void testFuselinkSend() {
    std::cout << "Starting fuselink_send test..." << std::endl;

    // Allocate test data
    uint64_t* d_data;
    uint64_t* h_data;
    CUDA_CHECK(hipMalloc(&d_data, TEST_DATA_SIZE));
    CUDA_CHECK(hipHostMalloc(&h_data, TEST_DATA_SIZE));

    // Initialize test data on GPU
    // dim3 block(256);
    // dim3 grid((TEST_DATA_SIZE / sizeof(uint64_t) + block.x - 1) / block.x);
    // initTestData<<<grid, block>>>(d_data, TEST_DATA_SIZE);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Create CUDA stream
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Test sending data
    std::cout << "Sending " << TEST_DATA_SIZE << " bytes..." << std::endl;
    int result = fuselink_send(d_data, TEST_DATA_SIZE, stream, 0);
    if (result != 0) {
        std::cerr << "fuselink_send failed with error code: " << result << std::endl;
        goto cleanup;
    }

    // Wait for send to complete
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Verify data was sent correctly
    // Note: In a real test, you would need to implement the receive side
    // and verify the data there. For now, we'll just check if the send completed.
    std::cout << "Send completed successfully" << std::endl;

cleanup:
    // Cleanup
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipHostFree(h_data));
}

int main() {
    // Enable CUDA printf
    // hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024 * 1024);
    
    // Run test
    testFuselinkSend();
    
    std::cout << "Test completed!" << std::endl;
    return 0;
}